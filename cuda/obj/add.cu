#include "hip/hip_runtime.h"
#include "add.cuh"

extern "C" {

#define cudaErrchk(ans) { GPUAssert((ans), __FILE__, __LINE__); }
inline void GPUAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ void add_array_d(double *a, double *b, double *c, long N) {
  long i = blockIdx.x * blockDim.x + threadIdx.x;  // an element_index
  if (i < N) { *(c+i) = *(a+i) + *(b+i); }
}


int add_array(double *a, double *b, double *c, long N) {
  
  double *a_d, *b_d, *c_d;

  size_t array_size_in_byte = N * sizeof(double);
  cudaErrchk( hipMalloc(&a_d, array_size_in_byte) );
  cudaErrchk( hipMalloc(&b_d, array_size_in_byte) );
  cudaErrchk( hipMalloc(&c_d, array_size_in_byte) );

  cudaErrchk( hipMemcpy(a_d, a, array_size_in_byte, hipMemcpyHostToDevice) );
  cudaErrchk( hipMemcpy(b_d, b, array_size_in_byte, hipMemcpyHostToDevice) );

  int block_size = 32;
  int grid_size = (N + block_size - 1) / block_size;
  add_array_d<<<grid_size,block_size>>>(a_d, b_d, c_d, N);

  cudaErrchk( hipMemcpy(c, c_d, array_size_in_byte, hipMemcpyDeviceToHost) );
  
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

  return 0;
}

}
